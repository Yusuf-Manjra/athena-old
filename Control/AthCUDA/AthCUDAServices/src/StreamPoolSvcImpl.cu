// Dear emacs, this is -*- c++ -*-
//
// Copyright (C) 2002-2020 CERN for the benefit of the ATLAS collaboration
//

// Local include(s).
#include "StreamPoolSvcImpl.h"

// AthCUDA include(s).
#include "AthCUDACore/Macros.cuh"
#include "AthCUDACore/StreamHolderHelpers.cuh"

// TBB include(s).
#include <tbb/concurrent_queue.h>

// System include(s).
#include <cassert>

namespace AthCUDA {

   class StreamPoolSvcImplData {

   public:
      /// Custom destructor, deleting the CUDA streams.
      ~StreamPoolSvcImplData() {
         while( ! m_streams.empty() ) {
            hipStream_t stream = nullptr;
            m_streams.pop( stream );
            assert( stream != nullptr );
            CUDA_IGNORE( hipStreamDestroy( stream ) );
         }
      }

      /// The concurrent pool of streams that @c StreamPoolSvcImpl manages
      tbb::concurrent_bounded_queue< hipStream_t > m_streams;

   }; // struct StreamPoolSvcImplData

   StreamPoolSvcImpl::StreamPoolSvcImpl( unsigned int nStreams,
                                         IStreamPoolSvc& svc )
   : m_svc( &svc ), m_data( new StreamPoolSvcImplData() ) {

      // Allocate the requested number of streams.
      for( unsigned int i = 0; i < nStreams; ++i ) {
         hipStream_t stream = nullptr;
         CUDA_EXP_CHECK( hipStreamCreate( &stream ) );
         m_data->m_streams.push( stream );
      }
   }

   StreamPoolSvcImpl::StreamPoolSvcImpl( StreamPoolSvcImpl&& parent )
   : m_svc( parent.m_svc ), m_data( parent.m_data ) {

      parent.m_data = 0;
   }

   StreamPoolSvcImpl::~StreamPoolSvcImpl() {

      // Delete the data object.
      if( m_data != nullptr ) {
         delete m_data;
      }
   }

   StreamPoolSvcImpl& StreamPoolSvcImpl::operator=( StreamPoolSvcImpl&& rhs ) {

      // Check if anything needs to be done.
      if( this == &rhs ) {
         return *this;
      }

      // Perform the move.
      m_svc  = rhs.m_svc;
      m_data = rhs.m_data;
      rhs.m_data = nullptr;

      // Return this object.
      return *this;
   }

   StreamHolder StreamPoolSvcImpl::getAvailableStream() {

      // A security check.
      assert( m_data != nullptr );

      // Get the next available stream for the user.
      hipStream_t stream = nullptr;
      m_data->m_streams.pop( stream );
      return StreamHolder( stream, m_svc );
   }

   void StreamPoolSvcImpl::yieldStream( StreamHolder& stream ) {

      // A security check.
      assert( m_data != nullptr );

      // Put the stream back into our queue.
      m_data->m_streams.push( getStream( stream ) );
      return;
   }

} // namespace AthCUDA
