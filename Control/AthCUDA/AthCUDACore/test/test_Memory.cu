#include "hip/hip_runtime.h"
// Dear emacs, this is -*- c++ -*-
//
// Copyright (C) 2002-2020 CERN for the benefit of the ATLAS collaboration
//

// Local include(s).
#include "AthCUDACore/Memory.cuh"

// System include(s).
#undef NDEBUG
#include <cassert>
#include <cmath>
#include <unistd.h>

int main() {

   {

      // Allocate some test arrays.
      auto array1 = AthCUDA::make_managed_array< float >( 500 );
      auto array2 = AthCUDA::make_managed_array< int >( 100 );
      auto array3 = AthCUDA::make_managed_array< double >( 1000 );

      // Make sure that they can be filled.
      for( int i = 0; i < 10; ++i ) {
         array1.get()[ i ] = 1.2f;
         array2.get()[ i ] = 123;
         array3.get()[ i ] = M_PI;
      }

   }
   // Wait for a moment, so that TBB would have time to schedule the deletion
   // of these arrays. Otherwise CUDA runtime may have already unloaded itself
   // by the time the hipFree(...) call is made.
   sleep( 1 );

   // Allocate some (device) arrays.
   AthCUDA::array< int > array4( 100 );
   AthCUDA::array< float > array5;
   array5.resize( 200 );

   // Make sure that we can write to these.
   for( int i = 0; i < 10; ++i ) {
      array4[ i ] = i;
      array5[ i ] = i * M_PI;
   }

   // Check that resizing works as intended.
   array5.resize( 100 );
   array5.resize( 500 );
   for( int i = 0; i < 10; ++i ) {
      assert( std::abs( array5[ i ] - i * M_PI ) < 0.0001 );
   }

   // Return gracefully.
   return 0;
}
