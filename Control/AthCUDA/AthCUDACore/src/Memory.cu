// Dear emacs, this is -*- c++ -*-
//
// Copyright (C) 2002-2021 CERN for the benefit of the ATLAS collaboration
//

// Local include(s).
#include "AthCUDACore/Memory.cuh"
#include "AthCUDACore/Info.h"
#include "AthCUDACore/Macros.cuh"
#include "AthCUDACore/TaskArena.h"

// TBB include(s).
#include <tbb/concurrent_queue.h>

// System include(s).
#include <cstdlib>
#include <stdexcept>
#include <thread>

namespace {

   /// Task used to delete pinned memory from the host
   class HostDeleterTask {

   public:
      /// Constructor with the memory pointer to delete
      HostDeleterTask( void* ptr ) : m_ptr( ptr ) {}

      /// Operator executing the functor
      void operator()() const {

         // Delete the memory.
         CUDA_EXP_CHECK( hipHostFree( m_ptr ) );
         return;
      }

   private:
      /// Pointer to delete (eventually)
#ifdef __clang__
      [[maybe_unused]]
#endif
      void* m_ptr;

   }; // class HostDeleterTask

   /// Task used to delete device memory
   class DeviceDeleterTask {

   public:
      /// Constructor with the memory pointer to delete
      DeviceDeleterTask( void* ptr ) : m_ptr( ptr ) {}

      /// Operator executing the functor
      void operator()() const {

         // Delete the memory.
         CUDA_EXP_CHECK( hipFree( m_ptr ) );
         return;
      }

   private:
      /// Pointer to delete (eventually)
#ifdef __clang__
      [[maybe_unused]]
#endif
      void* m_ptr;

   }; // class DeviceDeleterTask

} // private namespace

namespace AthCUDA {

   namespace details {

      void ManagedArrayDeleter::operator()( void* ptr ) {

         // Don't do anything for a null pointer.
         if( ptr == nullptr ) {
            return;
         }

         // If a device is available, then free up the memory using CUDA.
         if( Info::instance().nDevices() != 0 ) {
            taskArena().enqueue( ::DeviceDeleterTask( ptr ) );
            return;
         }

         // If not, then the memory was simply allocated with malloc...
         ::free( ptr );
         return;
      }

      void DeviceArrayDeleter::operator()( void* ptr ) {

         // Don't do anything for a null pointer.
         if( ptr == nullptr ) {
            return;
         }

         // If a device is available, then free up the memory using CUDA.
         if( Info::instance().nDevices() != 0 ) {
            taskArena().enqueue( ::DeviceDeleterTask( ptr ) );
            return;
         }

         // If not, then the memory was simply allocated with malloc...
         ::free( ptr );
         return;
      }

      void HostArrayDeleter::operator()( void* ptr ) {

         // Don't do anything for a null pointer.
         if( ptr == nullptr ) {
            return;
         }

         // If a device is available, then free up the memory using CUDA.
         if( Info::instance().nDevices() != 0 ) {
            taskArena().enqueue( ::HostDeleterTask( ptr ) );
            return;
         }

         // If not, then the memory was simply allocated with malloc...
         ::free( ptr );
         return;
      }

      /// Use CUDA managed memory if CUDA is available during the build and
      /// a CUDA device is available during runtime. Otherwise do the deed
      /// simply with standard C memory allocation.
      ///
      /// @param size The size of the array to create
      /// @return A pointer to the allocated array
      ///
      template< typename T >
      T* managedMallocHelper( std::size_t size ) {

         // For a zero sized array return a null pointer.
         if( size == 0 ) {
            return nullptr;
         }

         // The result pointer.
         T* result = 0;

         // Try to allocate the array in CUDA managed memory first.
         if( Info::instance().nDevices() != 0 ) {
            CUDA_EXP_CHECK( hipMallocManaged( &result, size * sizeof( T ) ) );
            return result;
         }

         // If that didn't work, fall back on simple malloc.
         result = static_cast< T* >( ::malloc( size * sizeof( T ) ) );
         return result;
      }

      /// Use CUDA device memory if CUDA is available during the build and
      /// a CUDA device is available during runtime. Otherwise do the deed
      /// simply with standard C memory allocation.
      ///
      /// @param size The size of the array to create
      /// @return A pointer to the allocated array
      ///
      template< typename T >
      T* deviceMallocHelper( std::size_t size ) {

         // For a zero sized array return a null pointer.
         if( size == 0 ) {
            return nullptr;
         }

         // The result pointer.
         T* result = 0;

         // Try to allocate the array in CUDA managed memory first.
         if( Info::instance().nDevices() != 0 ) {
            CUDA_EXP_CHECK( hipMalloc( &result, size * sizeof( T ) ) );
            return result;
         }

         // If that didn't work, fall back on simple malloc.
         result = static_cast< T* >( ::malloc( size * sizeof( T ) ) );
         return result;
      }

      /// Use CUDA to allocate page-locked memory on the host if CUDA is
      /// available, otherwise just allocate plain old memory.
      ///
      /// @param size The size of the array to create
      /// @return A pointer to the allocated array
      ///
      template< typename T >
      T* hostMallocHelper( std::size_t size ) {

         // For a zero sized array return a null pointer.
         if( size == 0 ) {
            return nullptr;
         }

         // The result pointer.
         T* result = 0;

         // Try to allocate the array in CUDA managed memory first.
         if( Info::instance().nDevices() != 0 ) {
            CUDA_EXP_CHECK( hipHostAlloc( &result, size * sizeof( T ),
                                           hipHostMallocDefault |
                                           hipHostMallocWriteCombined ) );
            return result;
         }

         // If that didn't work, fall back on simple malloc.
         result = static_cast< T* >( ::malloc( size * sizeof( T ) ) );
         return result;
      }

   } // namespace details

} // namespace AthCUDA

/// Helper macro for instantiating the allocator functions for different types
#define INST_MALLOC( TYPE )                                                    \
   template TYPE* AthCUDA::details::managedMallocHelper< TYPE >( std::size_t );\
   template TYPE* AthCUDA::details::hostMallocHelper< TYPE >( std::size_t );   \
   template TYPE* AthCUDA::details::deviceMallocHelper< TYPE >( std::size_t )

// Instantiate the array allocators for all "reasonable" primitive types.
INST_MALLOC( void* );
INST_MALLOC( char );
INST_MALLOC( unsigned char );
INST_MALLOC( short );
INST_MALLOC( unsigned short );
INST_MALLOC( int );
INST_MALLOC( unsigned int );
INST_MALLOC( long );
INST_MALLOC( unsigned long );
INST_MALLOC( long long );
INST_MALLOC( unsigned long long );
INST_MALLOC( float );
INST_MALLOC( double );

// Clean up.
#undef INST_MALLOC
