// Dear emacs, this is -*- c++ -*-
//
// Copyright (C) 2002-2020 CERN for the benefit of the ATLAS collaboration
//

// Local include(s).
#include "AthCUDACore/StreamHolderHelpers.cuh"

// Project include(s).
#include "AthCUDAInterfaces/StreamHolder.h"

namespace AthCUDA {

   hipStream_t getStream( StreamHolder&
#ifdef __HIPCC__
                                         holder
#endif // __HIPCC__
                                                ) {

#ifdef __HIPCC__
      return reinterpret_cast< hipStream_t >( holder.m_stream );
#else
      return nullptr;
#endif // __HIPCC__
   }

} // namespace AthCUDA
