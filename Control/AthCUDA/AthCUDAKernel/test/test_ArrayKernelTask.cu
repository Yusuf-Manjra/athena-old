#include "hip/hip_runtime.h"
// Dear emacs, this is -*- c++ -*-
//
// Copyright (C) 2002-2020 CERN for the benefit of the ATLAS collaboration
//

// Local include(s).
#include "AthCUDAKernel/ArrayKernelTask.cuh"

// AthCUDA include(s).
#include "AthCUDAInterfaces/StreamHolder.h"
#include "AthCUDACore/Info.h"
#include "AthCUDACore/Macros.cuh"

// System include(s).
#include <array>
#undef NDEBUG
#include <cassert>
#include <cmath>
#include <unistd.h>

/// Functor processing one variable
class Functor1 {
public:
   ATHCUDA_HOST_AND_DEVICE
   void operator()( std::size_t i, float* array1 ) {

      array1[ i ] *= 1.2f;
   }
}; // class Functor1

/// Functor processing two variables
class Functor2 {
public:
   ATHCUDA_HOST_AND_DEVICE
   void operator()( std::size_t i, float* array1, float multiplier,
                    double* array2 ) {

      array2[ i ] *= multiplier * array1[ i ];
   }
}; // class Functor2

/// Helper function initialising array elements
template< typename T, std::size_t ASIZE >
void initializeArray( std::array< T, ASIZE >& array );

int main() {

   // Make some arrays on the host.
   static const float MULTIPLIER = 1.34f;
   static const std::size_t ARRAY_SIZES = 100;
   std::array< float, ARRAY_SIZES > array1;
   std::array< double, ARRAY_SIZES > array2;

   // Allocate a CUDA stream for the test.
   hipStream_t stream = nullptr;
   if( AthCUDA::Info::instance().nDevices() > 0 ) {
      CUDA_EXP_CHECK( hipStreamCreate( &stream ) );
   }
   AthCUDA::StreamHolder sholder( stream, nullptr );

   // Test a kernel with just 1 array.
   AthCUDA::KernelStatus dummy;
   initializeArray( array1 );
   {
      auto kernel1 =
         AthCUDA::make_ArrayKernelTask< Functor1 >( dummy, ARRAY_SIZES,
                                                    array1.data() );
      kernel1->execute( sholder );

      // Wait for the task to finish.
      if( stream ) {
         CUDA_EXP_CHECK( hipDeviceSynchronize() );
      }
      kernel1->finished( 0, AthCUDA::IKernelTask::Asynchronous );
   }

   // Check that it produced the expected results.
   float maxError1 = 0;
   for( std::size_t i = 0; i < ARRAY_SIZES; ++i ) {
      maxError1 =
         std::max( maxError1,
                   std::abs( array1[ i ] -
                             static_cast< float >( i * 1.2f * 1.2f ) ) );
   }
   std::cout << "maxError1 = " << maxError1 << std::endl;
   assert( maxError1 < 0.001 );

   // Test a kernel with 2 arrays.
   initializeArray( array1 );
   initializeArray( array2 );
   {
      auto kernel2 =
         AthCUDA::make_ArrayKernelTask< Functor2 >( dummy, ARRAY_SIZES,
                                                    array1.data(), MULTIPLIER,
                                                    array2.data() );
      kernel2->execute( sholder );

      // Wait for the task to finish, and then destroy the CUDA stream.
      if( stream ) {
         CUDA_EXP_CHECK( hipDeviceSynchronize() );
         CUDA_EXP_CHECK( hipStreamDestroy( stream ) );
      }
      kernel2->finished( 0, AthCUDA::IKernelTask::Asynchronous );

   }

   // Check that it produced the expected results.
   double maxError2 = 0;
   for( std::size_t i = 0; i < ARRAY_SIZES; ++i ) {
      maxError2 =
         std::max( maxError2,
                   std::abs( array2[ i ] -
                             i * 1.2f * MULTIPLIER *
                             static_cast< double >( array1[ i ] ) ) );
   }
   std::cout << "maxError2 = " << maxError2 << std::endl;
   assert( maxError2 < 0.001 );

   // Wait for a moment, for all CUDA memory to be deleted, before terminating
   // the application.
   sleep( 1 );

   // Return gracefully.
   return 0;
}

template< typename T, std::size_t ASIZE >
void initializeArray( std::array< T, ASIZE >& array ) {

   for( std::size_t i = 0; i < ASIZE; ++i ) {
      array[ i ] = i * 1.2f;
   }
   return;
}
