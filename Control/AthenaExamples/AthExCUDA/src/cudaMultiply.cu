#include "hip/hip_runtime.h"
//
// Copyright (C) 2002-2020 CERN for the benefit of the ATLAS collaboration
//

// Local include(s).
#include "cudaMultiply.h"

// CUDA include(s).
#ifdef __HIPCC__
#   include <hip/hip_runtime.h>
#endif // __HIPCC__

// System include(s).
#include <iostream>

#ifdef __HIPCC__

/// Simple macro to run CUDA commands with
#define CUDA_CHECK( EXP )                                                     \
   do {                                                                       \
      const hipError_t ce = EXP;                                             \
      if( ce != hipSuccess ) {                                               \
         std::cerr << "Failed to execute: " << #EXP << std::endl;             \
         std::cerr << "Reason: " << hipGetErrorString( ce ) << std::endl;    \
         return;                                                              \
      }                                                                       \
   } while( false )

namespace AthCUDAExamples {

   /// Very simple kernel performing a multiplication on an array.
   __global__
   void cudaMultiplyKernel( int n, float* array, float multiplier ) {

      const int index = blockIdx.x * blockDim.x + threadIdx.x;
      if( index >= n ) {
         return;
      }

      array[ index ] *= multiplier;
      return;
   }

   /// GPU implementation of @c cudaMultiply
   void cudaMultiply( std::vector< float >& array, float multiplier ) {

      // If no CUDA device is available, complain.
      int nCudaDevices = 0;
      CUDA_CHECK( hipGetDeviceCount( &nCudaDevices ) );
      if( nCudaDevices == 0 ) {
         return;
      }

      // Allocate the array on the/a device, and copy the host array's content
      // to the device.
      float* deviceArray = nullptr;
      CUDA_CHECK( hipMalloc( &deviceArray, sizeof( float ) * array.size() ) );
      CUDA_CHECK( hipMemcpy( deviceArray, array.data(),
                              sizeof( float ) * array.size(),
                              hipMemcpyHostToDevice ) );

      // Run the kernel.
      static const int blockSize = 256;
      const int numBlocks = ( array.size() + blockSize - 1 ) / blockSize;
      cudaMultiplyKernel<<< numBlocks, blockSize >>>( array.size(),
                                                      deviceArray,
                                                      multiplier );
      CUDA_CHECK( hipDeviceSynchronize() );

      // Copy the array back to the host's memory.
      CUDA_CHECK( hipMemcpy( array.data(), deviceArray,
                              sizeof( float ) * array.size(),
                              hipMemcpyDeviceToHost ) );

      // Free the memory on the device.
      CUDA_CHECK( hipFree( deviceArray ) );
      return;
   }

} // namespace AthCUDAExamples

#else

namespace AthCUDAExamples {

   /// CPU implementation of @c cudaMultiply
   void cudaMultiply( std::vector< float >& array, float multiplier ) {

      for( float& element : array ) {
         element *= multiplier;
      }
   }

} // namespace AthCUDAExamples

#endif // __HIPCC__
